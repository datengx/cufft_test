#include "hip/hip_runtime.h"

#include "../common/book.h"
#include "./utils.h"
#include "./timing.h"
// #include "./cuda_kernels.cuh"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <complex>
#include <vector>
#include <cstring>


#define NX 128
#define NY 128
#define NZ 128
#define LX (2 * M_PI)
#define LY (2 * M_PI)
#define NUM_IMAGES 3


using namespace std;

typedef double     SimPixelType;

// __global__ void add_slices(PIXEL_TYPE* image_in, PIXEL_TYPE* image_out) {
//     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     int idx = tid % (128 * 128 * 2);
//     PIXEL_TYPE temp = image_in[tid];
//     // printf("%d\n", idx);
//     // if (tid < TOTAL_PIXELS) {
//     // if (tid < 16384) {
//     atomicAdd( &image_out[idx], temp );
//         // image_out[tid] = temp;
//     // }
//     // }
// }

 __global__ void Multiply_complex(SimPixelType* image_in, SimPixelType* image_in2) {
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     int idx = tid % (128 * 128 * 2);
     SimPixelType c1_real = image_in[tid*2];
	 SimPixelType c1_imag = image_in[tid*2+1];
	 SimPixelType c2_real = image_in2[tid*2];
	 SimPixelType c2_imag = image_in2[tid*2+1];
	 image_in[tid*2] = c1_real * c2_real - c1_imag * c2_imag;
	 image_in[tid*2+1] = c1_real * c2_imag + c1_imag * c2_real;
 }

int main() {
	hipDeviceReset();
	/* Create couple of images for testing */
	SimPixelType *x = new SimPixelType[NX * NY];
	SimPixelType *y = new SimPixelType[NX * NY];
	SimPixelType *in = new SimPixelType[NX * NY * NZ];
	/* A vector holding multiple images data */
	vector< SimPixelType* > image_vector;
	vector< SimPixelType* > dev_pointers_in;
	vector< SimPixelType* > dev_pointers_out;
	vector< SimPixelType* > imageOut_vector;
	vector< SimPixelType* > mult_image_vector;

	/* Create Fourier Kernel plan */
	hipfftHandle planr2c[NUM_IMAGES];
	hipfftHandle planc2r[NUM_IMAGES];

	/* Create an array of CUDA streams */
	hipStream_t streams_fft[NUM_IMAGES];

	/* Output image */
	complex<SimPixelType> *out = new complex<SimPixelType>[NX * NY * NZ];
	gpuErrchk( hipHostRegister( out, sizeof(SimPixelType)*NX*NY*NZ*2, hipHostRegisterPortable ) );
	// complex<SimPixelType>* out;
	// gpuErrchk( hipHostMalloc( &out, NX * NY * NZ * sizeof(SimPixelType) * 2 ) );
		/* Initialize it */
	memset( out, 0, sizeof(SimPixelType)*NX*NY*NZ*2 );

	/* Create the second argument image in the multiply kernel */
	SimPixelType* OTF = new SimPixelType[NX * NY * NZ * 2]; // Since the image is complex
	SimPixelType* dev_OTF;

	for (int p = 0; p < NZ; p++) {
		for(int j = 0; j < NY; j++) {
			for(int kk = 0; kk < NX; kk++) {
				OTF[(j * NX + kk) * 2] = kk + j;
				OTF[(j * NX + kk) * 2 + 1] = kk + j;
			}
		}
	}
	/* Reserve memory locations for the OTF image */
	gpuErrchk( hipMalloc( &dev_OTF, sizeof(SimPixelType)*NX*NY*NZ*2 ) );
	gpuErrchk( hipHostRegister( OTF, sizeof(SimPixelType)*NX*NY*NZ*2, hipHostRegisterPortable ) );

	for (unsigned i = 0; i < NUM_IMAGES; i++) {

		SimPixelType *vx = new SimPixelType[NX * NY * NZ];
		SimPixelType *mult_image = new SimPixelType[NX * NY * NZ * 2];
		// SimPixelType* vx;
		// hipHostMalloc( &vx, NX * NY * NZ * sizeof(SimPixelType) );
		for (int p = 0; p < NZ; p++) {
			for(int j = 0; j < NY; j++){
			    for(int kk = 0; kk < NX; kk++){
			        x[j * NX + kk] = kk * LX/NX;
			        y[j * NX + kk] = kk * LY/NY;

			        /* Put values in the new images */
			        vx[j * NX + kk + p * NX * NY] = cos(x[j * NX + kk] + y[j * NX + kk]);
			        if ( i == 0 ) {
			        	in[j * NX + kk + p * NX * NY] = cos(x[j * NX + kk] + y[j * NX + kk]);
			        }
			    }
			}
		}
		t1 = absoluteTime();
		gpuErrchk( hipHostRegister( vx, sizeof(SimPixelType)*NX*NY*NZ, hipHostRegisterPortable ) );
		gpuErrchk( hipHostRegister( mult_image, sizeof(SimPixelType)*NX*NY*NZ*2, hipHostRegisterPortable ) );
		t2 = absoluteTime();
  		std::cout << "\n\n Register time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;
		// for (int j = 0; j < NY; j++){
		//     for (int i = 0; i < NX; i++){
		//         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
		//         cout << vx[j * NX + i] << " ";
		//     }
		//     // printf("\n");
		//     cout << endl;
		// }
		// cout << endl;
		/* Allocate some spaces on the device */
		SimPixelType *d_vx;
		SimPixelType *d_out;
		/* Some space on the device */
		gpuErrchk(hipMalloc(&d_vx, NX * NY * NZ * sizeof(SimPixelType)));
		gpuErrchk(hipMalloc(&d_out, NX * NY * NZ * sizeof(hipfftDoubleComplex)));

		/* Create cufft FFT plans */
		int n[2] = {NX, NY};
		int inembed[] = {NX, NY};
		int onembed[] = {NX, NY};

		/* Forward Fourier Transform plan */
		hipfftPlanMany(&planr2c[i],
		            2, // rank
		            n, // dimension
		            inembed,
		            1, // istride
		            NX * NY, // idist
		            onembed,
		            1, //ostride
		            NX * NY, // odist
		            HIPFFT_D2Z,
		            NZ);



		/* Inverse Fourier Transform plan */
		hipfftPlanMany(&planc2r[i],
		            2, // rank
		            n, // dimension
		            onembed,
		            1, // istride
		            NX * NY, // idist
		            inembed,
		            1, //ostride
		            NX * NY, // odist
		            HIPFFT_Z2D,
		            NZ);

		cufftSetCompatibilityMode(planr2c[i], CUFFT_COMPATIBILITY_NATIVE);
		cufftSetCompatibilityMode(planc2r[i], CUFFT_COMPATIBILITY_NATIVE);
		/* Create streams associated with this 2 plans  */
		gpuErrchk( hipStreamCreate( &streams_fft[i] ));
		hipfftSetStream( planr2c[i], streams_fft[i] );
		// gpuErrchk( hipStreamCreate(&streams_ifft[i]) );
		// hipfftSetStream(&planc2r[i]);

		image_vector.push_back( vx );
		mult_image_vector.push_back( mult_image );
		dev_pointers_in.push_back( d_vx );
		dev_pointers_out.push_back( d_out );
	}


	/* Copying data to the device for processing */
	// hipMemcpy(d_vx, vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
	// hipMemcpy(d_out, out, NX * NY * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	t1 = absoluteTime();
	gpuErrchk( hipMemcpyAsync(
				dev_OTF,
				OTF,
				2*NX*NY*NZ*sizeof(SimPixelType),
				hipMemcpyHostToDevice,
				streams_fft[0]
	) );

	for (unsigned int j = 0; j < NUM_IMAGES; j++ ) {
		gpuErrchk( hipMemcpyAsync( dev_pointers_in[j],
									image_vector[j],
									NX*NY*NZ*sizeof(SimPixelType),
									hipMemcpyHostToDevice,
									streams_fft[j]) );
		gpuErrchk( hipMemcpyAsync( dev_pointers_out[j],
									out,
									NX*NY*NZ*sizeof(hipfftDoubleComplex),
									hipMemcpyHostToDevice,
									streams_fft[j] ) );

	}

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		hipfftExecD2Z( planr2c[j],
					  (SimPixelType*)dev_pointers_in[j],
					  (hipfftDoubleComplex*)dev_pointers_out[j]);
		Multiply_complex<<< NX*NY*NZ/512, 512, 0, streams_fft[j] >>>( dev_pointers_out[j],
						  dev_OTF
							);
	}

	t2 = absoluteTime();
  	std::cout << "\n\n Streaming time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;
//	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
//		hipfftSetStream(planc2r[j], streams_fft[j]);
//	}
//
//	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
//		hipfftExecZ2D( planc2r[j], (hipfftDoubleComplex*)dev_pointers_out[j], (SimPixelType*)dev_pointers_in[j]);
//	}

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipMemcpyAsync( mult_image_vector[j], dev_pointers_out[j], 2*NX*NY*NZ*sizeof(SimPixelType), hipMemcpyDeviceToHost, streams_fft[j] ) );
	}

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipStreamSynchronize( streams_fft[j] ) );
	}
	t1 = absoluteTime();
	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipHostUnregister(image_vector[j]) );
		gpuErrchk( hipHostUnregister(mult_image_vector[j]) );
		// gpuErrchk( hipHostFree(image_vector[j]) );
	}
	gpuErrchk( hipHostUnregister(OTF) );
	gpuErrchk( hipHostUnregister(out) );
	// gpuErrchk( hipHostFree( out ) );
	t2 = absoluteTime();
  	std::cout << "\n\n Host Unregister time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;

	/* Cast into complex value array */
	complex< SimPixelType >* complex_array = reinterpret_cast< complex< SimPixelType >* >( mult_image_vector[1] );

   	for (int j = 0; j < NY; j++){
	     for (int i = 0; i < NX; i++){
	         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
	         // SimPixelType* vx = image_vector[1];
//	         cout << image_vector[0][j * NX + i]/( NX * NY ) << " ";
			cout << complex_array[j * NX + i] << " ";
	     }
	     // printf("\n");
	     cout << endl;
	 }
	// cout << endl;
	// for (int j = 0; j < NY; j++){
	//     for (int i = 0; i < NX; i++){
	//         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
	//         cout << in[j * NX + i] << " ";
	//     }
	//     // printf("\n");
	//     cout << endl;
	// }

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipFree( dev_pointers_in[j] ) );
		gpuErrchk( hipFree( dev_pointers_out[j] ) );
		hipStreamDestroy( streams_fft[j] );
		delete[] image_vector[j];
		delete[] mult_image_vector[j];
	}
	gpuErrchk( hipFree( dev_OTF ) );
	delete[] OTF;
	delete[] out;
	delete[] x;
	delete[] y;

	hipDeviceReset();

	// hipfftPlan2d(&planr2c, NY, NX, HIPFFT_D2Z);
	// hipfftPlan2d(&planc2r, NY, NX, HIPFFT_Z2D);

	// hipfftExecD2Z(planr2c, (hipfftDoubleReal *)d_vx, (hipfftDoubleComplex *)d_out);
	// hipfftExecZ2D(planc2r, (hipfftDoubleComplex *)d_out, (hipfftDoubleReal *)d_vx);


	/* Copy results back from the device */
	// hipMemcpy(vx, d_vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);







    // int count = 0;
    // hipDeviceProp_t prop;
    // int dev_id;
    // //  determining how many devices are available to use on the computer
    // HANDLE_ERROR( hipGetDeviceCount( &count ) );
    // printf("There are %d device(s) on this computer.\n", count);
    // // Iterates through each of the device on this computer

    // printDevInfo(count, prop);



	return 0;
}
#pragma clang diagnostic pop