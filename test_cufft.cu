#include "hip/hip_runtime.h"

#include "../common/book.h"
#include "./utils.h"
// #include "./cuda_kernels.cuh"
#include <hipfft/hipfft.h>
#include <iostream>
#include <complex>


#define NX 32
#define NY 32
#define LX (2 * M_PI)
#define LY (2 * M_PI)

using namespace std;

typedef double     SimPixelType;

int main() {


	SimPixelType *x = new SimPixelType[NX * NY];
	SimPixelType *y = new SimPixelType[NX * NY];
	SimPixelType *vx = new SimPixelType[NX * NY];
	complex<SimPixelType> *out = new complex<SimPixelType>[NX * NY];
	for(int j = 0; j < NY; j++){
	    for(int i = 0; i < NX; i++){
	        x[j * NX + i] = i * LX/NX;
	        y[j * NX + i] = j * LY/NY;
	        vx[j * NX + i] = cos(x[j * NX + i] + y[j * NX + i]);
	    }
	}

	for (int j = 0; j < NY; j++){
	    for (int i = 0; i < NX; i++){
	        // printf("%.3f ", vx[j*NX + i]/(NX*NY));
	        cout << vx[j * NX + i] << " ";
	    }
	    // printf("\n");
	    cout << endl;
	}
	cout << endl;
	SimPixelType *d_vx;
	SimPixelType *d_out;
	hipMalloc(&d_vx, NX * NY * sizeof(SimPixelType));
	hipMalloc(&d_out, NX * NY * sizeof(hipfftDoubleComplex));
	hipMemcpy(d_vx, vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, NX * NY * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

	hipfftHandle planr2c;
	hipfftHandle planc2r;

	int n[2] = {NX, NY};
	int inembed[] = {NX, NY};
	int onembed[] = {NX, NY};
	int depth = 128;

	/* Forward Fourier Transform plan */
	hipfftPlanMany(&planr2c,
	            2, // rank
	            n, // dimension
	            inembed,
	            1, // istride
	            NX * NY, // idist
	            onembed,
	            1, //ostride
	            NX * NY, // odist
	            HIPFFT_D2Z,
	            1);


	
	/* Inverse Fourier Transform plan */
	hipfftPlanMany(&planc2r,
	            2, // rank
	            n, // dimension
	            onembed,
	            1, // istride
	            NX * NY, // idist
	            inembed,
	            1, //ostride
	            NX * NY, // odist
	            HIPFFT_Z2D,
	            1);







	// hipfftPlan2d(&planr2c, NY, NX, HIPFFT_D2Z);
	// hipfftPlan2d(&planc2r, NY, NX, HIPFFT_Z2D);
	cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE);
	cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE);
	hipfftExecD2Z(planr2c, (hipfftDoubleReal *)d_vx, (hipfftDoubleComplex *)d_out);
	hipfftExecZ2D(planc2r, (hipfftDoubleComplex *)d_out, (hipfftDoubleReal *)d_vx);
	hipMemcpy(vx, d_vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);


	for (int j = 0; j < NY; j++){
	    for (int i = 0; i < NX; i++){
	        // printf("%.3f ", vx[j*NX + i]/(NX*NY));
	        cout << vx[j * NX + i]/( NX * NY) << " ";
	    }
	    // printf("\n");
	    cout << endl;
	}

	return 0;
}